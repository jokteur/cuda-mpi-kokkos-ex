#include "hip/hip_runtime.h"
// #include <Kokkos_Core.hpp>
#include <iostream>
#include <mpi.h>
#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define BLOCKSIZE 1024

void checkCUDAError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void times2(double* a, double* b) {
    int i = blockIdx.x;
    b[i] = 2 * a[i];
}

void loop() {
    size_t num_iterations = 500000;
    int mpi_rank, mpi_size;

    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);

    // Set cuda device from mpi rank
    hipSetDevice(mpi_rank);
    hipStream_t stream;
    hipStreamCreate(&stream);

    size_t N = 4194304;

    double* A = (double*)malloc(sizeof(double) * N);
    double* B = (double*)malloc(sizeof(double) * N);
    for (size_t i = 0; i < N; i++) {
        A[i] = 1.5;
        B[i] = 0;
    }

    double* dA, * dB;
    hipMalloc(&dA, sizeof(double) * N); checkCUDAError("Error allocating dA");
    hipMemcpy(dA, A, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Error copying A");
    hipMalloc(&dB, sizeof(double) * N); checkCUDAError("Error allocating dB");
    hipMemcpy(dB, B, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Error copying B");


    if (mpi_rank == 0)
        std::cout << "===== Working parallel ===== " << std::endl;

    auto t1 = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < num_iterations;i++) {
        times2 << <N / BLOCKSIZE, BLOCKSIZE, 0, stream >> > (dA, dB);
        times2 << <N / BLOCKSIZE, BLOCKSIZE, 0, stream >> > (dB, dA);
    }
    hipDeviceSynchronize();
    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "From rank:" << mpi_rank << ", time to execution: " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() / 1000.0 << std::endl;

    MPI_Barrier(MPI_COMM_WORLD);

    if (mpi_rank == 0)
        std::cout << "===== Bugged parallel (local sync) ===== " << std::endl;

    t1 = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < num_iterations;i++) {
        times2 << <N / BLOCKSIZE, BLOCKSIZE, 0, stream >> > (dA, dB);
        times2 << <N / BLOCKSIZE, BLOCKSIZE, 0, stream >> > (dB, dA);
        hipStreamSynchronize(stream);
    }
    t2 = std::chrono::high_resolution_clock::now();
    std::cout << "From rank:" << mpi_rank << ", time to execution: " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() / 1000.0 << std::endl;
    // Kokkos::fence();
    hipFree(dA);
    hipFree(dB);
    free(A);
    free(B);
}